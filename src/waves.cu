#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

using namespace cv;

struct BGR {
  uint8_t blue;
  uint8_t green;
  uint8_t red;
};

__device__ void pixel_func(int x, int y, BGR *out) {
  double t = pow(1.0 - fabs(sin((double)y / 20.0)), 1.5);
  uint8_t v = (uint8_t)(t * 255.0);
  out->blue = out->green = out->red = v;
}

__global__
void texture_kernel(int dim_x, int dim_y, BGR *out) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  BGR *elt = out + (y * dim_x + x);
  pixel_func(x, y, elt);
}

int main(int argc, char** argv )
{
  if ( argc != 2 )
  {
    printf("usage: DisplayImage.out <Image_Path>\n");
    return -1;
  }
  size_t dim = 512;
  Mat texture(dim, dim, CV_8UC3);

  for (int i = 0; i < texture.rows; i++) {
    for (int j = 0; j < texture.cols; j++) {
      BGR &p = texture.ptr<BGR>(i)[j];
      p.blue = p.green = p.red = 0;
    }
  }

  printf("total: %ld\n", texture.total());
  printf("elem: %ld\n", texture.elemSize());

  BGR *d_texture;
  size_t texture_bytes = texture.total() * texture.elemSize();
  hipMalloc((void **)&d_texture, texture_bytes);
  // TODO this can probably be skipped
  hipMemcpy(d_texture, texture.data, texture_bytes, hipMemcpyHostToDevice);

  int thrd_x = 8;
  int thrd_y = 8;
  dim3 grid(ceil(dim/(double)thrd_x), ceil(dim/(double)thrd_y), 1);
  dim3 thrd(thrd_x, thrd_y, 1);

  texture_kernel<<<grid, thrd>>>(dim, dim, d_texture);

  hipMemcpy(texture.data, d_texture, texture_bytes, hipMemcpyDeviceToHost);

  // TODO write the final texture to texture memory and render directly with OpenGL
  namedWindow("Display Image", WINDOW_AUTOSIZE );
  imshow("Display Image", texture);
  imwrite(argv[1], texture);
  waitKey(0);
  return 0;
}
